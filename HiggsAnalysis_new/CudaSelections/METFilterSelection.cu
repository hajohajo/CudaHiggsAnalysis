
#include <hip/hip_runtime.h>
//
//  METFilterSelection.cpp
//  HiggsAnalysis_new
//
//  Created by Joona Havukainen on 5/31/19.
//  Copyright © 2019 Joona Havukainen. All rights reserved.
//

__global__
void metFilterSelection(float *inputArray, bool *passedArray, int variablesPerEvent, int nEvents, int metFilterIndex)
{
    int processIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int localIndex = processIndex * variablesPerEvent;
    
    int nMETFilters = 7;
    if(processIndex<nEvents)
    {
        passedArray[processIndex*nMETFilters + 0] = inputArray[localIndex + metFilterIndex + 0];
        passedArray[processIndex*nMETFilters + 1] = inputArray[localIndex + metFilterIndex + 1];
        passedArray[processIndex*nMETFilters + 2] = inputArray[localIndex + metFilterIndex + 2];
        passedArray[processIndex*nMETFilters + 3] = inputArray[localIndex + metFilterIndex + 3];
        passedArray[processIndex*nMETFilters + 4] = inputArray[localIndex + metFilterIndex + 4];
        passedArray[processIndex*nMETFilters + 5] = inputArray[localIndex + metFilterIndex + 5];
        passedArray[processIndex*nMETFilters + 6] = inputArray[localIndex + metFilterIndex + 6];

    }
    
}
