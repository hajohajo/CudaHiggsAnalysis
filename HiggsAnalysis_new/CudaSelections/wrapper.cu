//
//  wrapper.cpp
//  HiggsAnalysis_new
//
//  Created by Joona Havukainen on 5/28/19.
//  Copyright © 2019 Joona Havukainen. All rights reserved.
//

#include <iostream>
#include "wrapper.h"
#include "TauSelection.cuh"

void wrapper(float *array, int entries, int nVariables)
{
    float *d_array;
    hipMalloc(&d_array, nVariables*entries*sizeof(float));
    hipMemcpy(d_array, array, nVariables*entries*sizeof(float), hipMemcpyHostToDevice);
    
    int blocks = (100000+1024)/1024;
    tauSelection<<<blocks, 1024>>>(d_array, entries);

    std::cout<<"Selection done"<<std::endl;
    hipFree(d_array);
}
